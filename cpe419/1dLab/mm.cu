#include "hip/hip_runtime.h"
/* Alexander DeMello
 * CPE 419 lab 01 v3
 * Matrix Multiplication using CUDA
 */

#include "mm.h"

int main (int argc, char*argv[]) {
   int resultRow, resultCol;
   int *f1rows, *f1cols, *f2rows, *f2cols;
   FILE *inFile1, *inFile2;
   myValue* result;

   //open files to be read into 2d arrays
   inFile1 = fopen(argv[1], "rb");
   inFile2 = fopen(argv[2], "rb");

   f1rows = (int*)malloc(sizeof(int));
   f1cols = (int*)malloc(sizeof(int));
   f2rows = (int*)malloc(sizeof(int));
   f2cols = (int*)malloc(sizeof(int));
   //read the values of rows/col of file 1/2 and store them
   fread(f1rows, 4, 1, inFile1);
   fread(f1cols, 4, 1, inFile1);
   fread(f2rows, 4, 1, inFile2);
   fread(f2cols, 4, 1, inFile2);

   resultRow = *f1rows;
   resultCol = *f2cols;

   if((*f1cols) != (*f2rows)) {
      printf("Cannot multiply these matrices");
      exit(0);
   }

   //call mmRead and return result array (call compute in mmRead)
   result = mmRead(inFile1, inFile2, f1rows, f1cols, f2rows, f2cols);

   //call mmWrite to write result array to output file
   mmWrite(result, resultRow, resultCol);
   
   //free memory and close files
   free(f1rows);
   free(f1cols);
   free(f2rows);
   free(f2cols);
   
   return 0;
}

void mmWrite(myValue *result, int resultRow, int resultCol) {
   FILE *output;
   int i, j;
   
   output = fopen("result.out", "w");
   //write result matrix to an output file
   for(i =0; i < resultRow; i++) {
      for(j = 0; j < resultCol; j++) {
         fprintf(output, "%.2f ", *(result + (i*resultCol + j)));
      }
      fprintf(output, "\n");
   }
   fclose(output);
   free(result);
}


myValue* mmRead(FILE *inFile1, FILE*inFile2, int* f1rows, int* f1cols,
      int* f2rows, int* f2cols) {
   myValue *mat1H, *mat2H, *result;
   int i, j;
   
   //allocate memory for storing infile values
   mat1H = (myValue*)calloc((*f1rows) * (*f1cols), sizeof(myValue));
   
   mat2H = (myValue*)calloc((*f2rows) * (*f2cols), sizeof(myValue));

   //read matrices into now allocated 2d arrays in row major
   for(i = 0; i < (*f1rows); i++) {
      for(j = 0; j < (*f1cols); j++) {
         fread(mat1H + (i * (*f1cols) + j), 4, 1, inFile1);
      }
   }

   //second matrix in row major
   for(i = 0; i < (*f2rows); i++) {
      for(j = 0; j < (*f2cols); j++) {
         fread(mat2H + (i * (*f2cols) + j), 4, 1, inFile2);
      }
   }
   
   //close files that are no longer needed
   fclose(inFile1);
   fclose(inFile2);

   //call mmcompute after some setup for outside help
   result = (myValue*)calloc((*f1rows) * (*f2cols), sizeof(myValue));
   mmGpuSetup(mat1H, mat2H, result, *f1rows, *f1cols, *f2rows, *f2cols);

   return result;

}

void mmGpuSetup(myValue* mat1H, myValue* mat2H, myValue* resultH,
      int m, int n, int p, int q) {
   myValue *resultD, *mat1D, *mat2D;

   //allocate result matrix memory on host and device
   //device
   hipMalloc((void **)&resultD, m * q * sizeof(myValue));
   hipMalloc((void **)&mat1D, m * n * sizeof(myValue));
   hipMalloc((void **)&mat2D, p * q * sizeof(myValue));

   //put host memory on device to be calculated
   hipMemcpy(mat1D, mat1H, m * n * sizeof(myValue), hipMemcpyHostToDevice);
   hipMemcpy(mat2D, mat2H, p * q * sizeof(myValue), hipMemcpyHostToDevice);
   hipMemcpy(resultD, resultH, m*q*sizeof(myValue), hipMemcpyHostToDevice);

   //calculate using threads on GPU
   //setup kernel call
   //matrix multiply cuda

   dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
   dim3 dimGrid((q + dimBlock.x -1)/dimBlock.x, (m + dimBlock.y -1)/dimBlock.y);
   mmCompute<<<dimGrid, dimBlock>>>(mat1D, mat2D, resultD, m, n, p, q);
   
   //bring back memory to host and free gfx mem
   hipMemcpy(resultH, resultD, m*q*sizeof(myValue), hipMemcpyDeviceToHost);
   hipFree(mat1D);
   hipFree(mat2D);
   hipFree(resultD);

}

__global__ void mmCompute(myValue* mat1D, myValue* mat2D, myValue* resultD,
      int m, int n, int p, int q) {
   int row, col, k;
   myValue sum;
   row = blockIdx.y * blockDim.y + threadIdx.y;
   col = blockIdx.x * blockDim.x + threadIdx.x;
   sum = 0;

   //calculate using threads on GPU
   for(k = 0; k < n; ++k) {
      sum += mat1D[row * n + k] * mat2D[k * q + col];
   }
   resultD[row * q + col] = sum;
}
      















