#include "hip/hip_runtime.h"
#include <stdlib.h>
#include "cudacuda.h"
#include "hip/hip_runtime.h"
#include <hipblas.h>
#include "hip/hip_runtime_api.h"

XYZ *cuLocs;
XYZ *cuVels;
double *cuMass;

XYZ *cuForces;

int partCount;

double parth, parte2;

dim3 sblocks;
dim3 sthreads;
dim3 fblocks;
dim3 fthreads;

__global__ void stepParticles(XYZ *locs, XYZ *vels, XYZ *forces, double *mass, double h, int numParts) {
    int i = blockIdx.x * 1024 + threadIdx.x;

    if (i < numParts) {
      XYZ *l = locs + i;
      XYZ *v = vels + i;
      XYZ f = forces[i];
      double m = mass[i];
    
      v->x += h * f.x / m;
      v->y += h * f.y / m;
      v->z += h * f.z / m;
    
      l->x += h * v->x;
      l->y += h * v->y;
      l->z += h * v->z;
    }
}

__global__ void calculateForces(XYZ *locs, XYZ *forces, double *mass, double e2, int numParts)
{
  int tr = blockIdx.x * 1024 + threadIdx.x;

  int i = tr / numParts;
  int j = tr % numParts;
  
  //for (int i = 0; i < parts.size(); i++)
  //for (int j = i + 1; j < parts.size(); j++)
  if (i < j && i < numParts && j < numParts) {
    double massMult = mass[i] * mass[j];
    XYZ l1 = locs[i];
    XYZ l2 = locs[j];
    XYZ disp;
    disp.x = l2.x - l1.x;
    disp.y = l2.y - l1.y;
    disp.z = l2.z - l1.z;
    double dispNorm = sqrt(pow(disp.x, 2) + pow(disp.y, 2) + pow(disp.z, 2));

    double calc = massMult / sqrt(pow(pow(dispNorm, 2) + e2, 3));

    XYZ *f1 = forces + i;
    XYZ *f2 = forces + j;
    f1->x += calc * disp.x;
    f1->y += calc * disp.y;
    f1->z += calc * disp.z;

    f2->x += calc * -disp.x;
    f2->y += calc * -disp.y;
    f2->z += calc * -disp.z;
  }
}

void cudaStepParticles(XYZ *locs) {
  checkCudaErrors(hipMemset(cuForces, 0, sizeof(XYZ) * partCount));  

  calculateForces<<<fblocks, fthreads>>>(cuLocs, cuForces, cuMass, parte2, partCount);
  stepParticles<<<sblocks, sthreads>>>(cuLocs, cuVels, cuForces, cuMass, parth, partCount);
  checkCudaErrors(hipMemcpy(locs, cuLocs, sizeof(XYZ) * partCount, hipMemcpyDeviceToHost)); 
}


void cudaCleanup() {
  checkCudaErrors(hipFree(cuLocs));
  checkCudaErrors(hipFree(cuVels));
  checkCudaErrors(hipFree(cuMass));
  checkCudaErrors(hipFree(cuForces));
}

void cudaInitParticles(XYZ *locs, XYZ *vels, double *mass, 
                            int numParts, double hin, double e2in) {
  partCount = numParts;
  parth = hin;
  parte2 = e2in;
  int tCount = numParts * (numParts + 1) / 2;

  dim3 fthreadMake(1024);
  fthreads = fthreadMake;
  dim3 fblockMake(tCount / 1024 + 1);
  fblocks = fblockMake;

  dim3 sthreadMake(1024);
  sthreads = sthreadMake;
  dim3 sblockMake(numParts * numParts / 1024 + 1);
  sblocks = sblockMake;

  checkCudaErrors(hipMalloc((void **) &cuLocs, sizeof(XYZ) * numParts));
  checkCudaErrors(hipMemcpy(cuLocs, locs, sizeof(XYZ) * numParts, hipMemcpyHostToDevice));
  
  checkCudaErrors(hipMalloc((void **) &cuVels, sizeof(XYZ) * numParts));
  checkCudaErrors(hipMemcpy(cuVels, vels, sizeof(XYZ) * numParts, hipMemcpyHostToDevice));

  checkCudaErrors(hipMalloc((void **) &cuMass, sizeof(double) * numParts));
  checkCudaErrors(hipMemcpy(cuMass, mass, sizeof(double) * numParts, hipMemcpyHostToDevice));
  
  // will get set to 0 at start of actual run
  checkCudaErrors(hipMalloc((void **) &cuForces, sizeof(XYZ) * numParts));
}




